#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include "dnn.hpp"

using namespace std;

//Define parameters for the two convolution cases
#define Ni 64
#define Nn 64

#define Kx 3
#define Ky 3

#define Nx 224
#define Ny 224


//Define the parameters if not defined externally
#ifndef Sy
#define Sy 1
#define Sx 1
#endif

#ifndef Tnn
//Tiling Sizes
#define Tnn 32
#define Tn  16
#define Ti  16

#define Ty  8
#define Tx  8
#endif

#define NYPAD (Ny+Ky)
#define NXPAD (Nx+Kx)

#define NYSCL (Ny/Sy)
#define NXSCL (Nx/Sx)

#define SYNAPSE_SIZE (1L*Ky*Kx*Nn*Ni)

VTYPE (*synapse)[Ky][Kx][Nn][Ni];
VTYPE (*neuron_i)[NYPAD][NXPAD][Ni];
VTYPE (*neuron_n)[NYSCL][NXSCL][Nn];
VTYPE (*neuron_n2)[NYSCL][NXSCL][Nn];


VTYPE (*cuda_result)[NYSCL][NXSCL][Nn]; // memory to hold cuda result

void fill_convolution_shared_simple(VTYPE (&synapse)[Ky][Kx][Nn][Ni], 
                                    VTYPE (&neuron_i)[NYPAD][NXPAD][Ni]) {
  for(int yy = 0; yy < Ky; ++yy) {
    for(int xx = 0; xx < Kx; ++xx) {
      for(int nn = 0; nn < Nn; ++nn) {
        for(int ni = 0; ni < Ni; ++ni) {
          synapse[yy][xx][nn][ni] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
        } } } }
  for(int yy = 0; yy < NYPAD; ++yy) {
    for(int xx = 0; xx < NXPAD; ++xx) {      
      for(int ni = 0; ni < Ni; ++ni) {
        neuron_i[yy][xx][ni] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
  }  }  }
}

std::pair<int,int> convolution_layer_blocked(
                                  VTYPE (&synapse)[Ky][Kx][Nn][Ni],
                                  VTYPE (&neuron_i)[NYPAD][NXPAD][Ni],
                                  VTYPE (&neuron_n)[NYSCL][NXSCL][Nn]){
  //int c1=0,c2=0;
  VTYPE sum[Nn]={0};

  for (int yy = 0; yy < Ny; yy += Ty) {
    for (int xx = 0; xx < Nx; xx += Tx) {
      for (int nnn = 0; nnn < Nn; nnn += Tnn) {
        int yout = yy/Sy;
        for (int y = yy; y < yy + Ty; y += Sy) { // tiling for y;
          int xout = xx/Sx;

          for (int x = xx; x < xx + Tx; x += Sx) { // tiling for x;

            for (int nn = nnn; nn < nnn + Tnn; nn += Tn) {
              for (int n = nn; n < nn + Tn; n++) {
                sum[n] = 0;
              }

              for (int ky = 0; ky < Ky; ky++) {  // sliding window;
                for (int kx = 0; kx < Kx; kx++) {

                  int ii = 0;
                  VTYPE sum_sc;

                  for (; ii < Ni -Ti+1; ii += Ti) {
                    for (int n = nn; n < nn + Tn; n++) {
                      sum_sc=0;
                      for (int i = ii; i < ii + Ti; i++) {
                        VTYPE sv = synapse[ky][kx][n][i];
                        VTYPE nv = neuron_i[ky + y][kx + x][i];
                        sum_sc+=sv*nv;
                      }
                      sum[n]+=sum_sc;
                    }
                  }
                }
              }

              //transfer
              for (int n = nn; n < nn + Tn; n++) {
                neuron_n[yout][xout][n] = transfer(sum[n]);
              }
            }
            xout++; 
          }
          yout++;
        }
      }
    }
  }
}

void  convolution_layer(VTYPE (&synapse)[Ky][Kx][Nn][Ni], 
                               VTYPE (&neuron_i)[NYPAD][NXPAD][Ni], 
                               VTYPE (&neuron_n)[NYSCL][NXSCL][Nn]){
  VTYPE sum[Nn]={0};

  // — Original code — (excluding nn, ii loops)
  int yout = 0;
  for (int y = 0; y < Ny; y += Sy) { // tiling for y;
    int xout = 0;
    for (int x = 0; x < Ny; x += Sx) { // tiling for x;
      for (int nn = 0; nn < Nn; nn += Tn) {
        for (int n = nn; n < nn + Tn; n++) {
          sum[n]=0;
        }

        // sliding window;
        for (int ky = 0; ky < Ky; ky++)
          for (int kx = 0; kx < Kx; kx++)
            for (int n = nn; n < nn + Tn; n++)
              for (int i = 0; i < Ni; i++) {
                VTYPE sv = synapse[ky][kx][n][i];
                VTYPE nv = neuron_i[ky + y][kx + x][i];
                sum[n]+=sv*nv;
              }
        for (int n = nn; n < nn + Tn; n++) {
          neuron_n[yout][xout][n] = transfer(sum[n]);
        }
      }
      xout++; 
    }
    yout++;
  }
}

//cuda convolution function
__global__ void conv_cu(    VTYPE(&synapse)[Nn][Ni][Ky][Kx],
                            VTYPE(&neuron_i)[Ni][NYPAD][NXPAD],
                            VTYPE(&neuron_n)[Nn][NYSCL][NXSCL]
                        )
{
    if(blockIdx.x*1024+threadIdx.x < (Ny*Nx))
    {
        int ix = ((blockIdx.x * 1024) + threadIdx.x) % Nx;
        int iy = ((blockIdx.x * 1024) + threadIdx.x) / Nx;
        
        for(int out=0;out<Nn;out++)
        {
            VTYPE sum = 0;
            for(int y=iy;y<iy+3;y++)
            {
                for(int x=ix;x<ix+3;x++)
                {
                    for(int in=0;in<Ni;in++)
                    {
                        sum += neuron_i[in][y][x] * synapse[out][in][y-iy][x-ix];
                    }
                }
            }


	
            if(sum < 0)
            {
                neuron_n[out][iy][ix] = sum/4;
            }
            else
            {
                neuron_n[out][iy][ix] = sum;
            }

        }
    }
    
}


int main(const int argc, const char** argv) {
  
    cout << "allocating memory\n";
    synapse   = (VTYPE (*)[Ky][Kx][Nn][Ni])aligned_malloc(64,SYNAPSE_SIZE*sizeof(VTYPE));
    neuron_i  = (VTYPE (*)[NYPAD][NXPAD][Ni])aligned_malloc(64,NYPAD*NXPAD*Ni*sizeof(VTYPE));
    neuron_n  = (VTYPE (*)[NYSCL][NXSCL][Nn])aligned_malloc(64,NYSCL*NXSCL*Nn*sizeof(VTYPE));
    neuron_n2 = (VTYPE (*)[NYSCL][NXSCL][Nn])aligned_malloc(64,NYSCL*NXSCL*Nn*sizeof(VTYPE));

    //declare memory for cuda arrays
    VTYPE(*synapse_cu)[Nn][Ni][Ky][Kx];
    VTYPE(*neuron_i_cu)[Ni][NYPAD][NXPAD];
    VTYPE(*neuron_n_cu)[Nn][NYSCL][NXSCL];
    
    //memory for result since cuda array dimensions are different
    
    cuda_result = (VTYPE (*)[NYSCL][NXSCL][Nn])aligned_malloc(64,NYSCL*NXSCL*Nn*sizeof(VTYPE));
    
    //allocate memory for cuda arrays
    hipMallocManaged(&synapse_cu,SYNAPSE_SIZE*sizeof(VTYPE));
    hipMallocManaged(&neuron_i_cu,NYPAD*NXPAD*Ni*sizeof(VTYPE));
    hipMallocManaged(&neuron_n_cu,NYSCL*NXSCL*Nn*sizeof(VTYPE));
    
    cout << "initializing arrays\n";
    fill_convolution_shared_simple(*synapse,*neuron_i);

    //copy to memory allocated for cuda
    //must do manually since dimensions are different
    
    //neuron input
    for(int i=0;i<NYPAD;i++)
    {
        for(int j=0;j<NXPAD;j++)
        {
            for(int in=0;in<Ni;in++)
            {
                (*neuron_i_cu)[in][i][j] = (*neuron_i)[i][j][in];
            }
        }
    }
    cout << "neuron input copied\n";
    
    //synapse
    for(int i=0;i<Ky;i++)
    {
        for(int j=0;j<Kx;j++)
        {
            for(int in=0;in<Ni;in++)
            {
                for(int out=0;out<Nn;out++)
                {
                    (*synapse_cu)[in][out][i][j] = (*synapse)[i][j][in][out];
                }
            }
        }
    }
    cout << "synapse copied\n";
    
    cout << "starting computation\n";
    //Simple Version
    begin_roi();
    convolution_layer(*synapse,*neuron_i,*neuron_n);
    end_roi();
    cout << "simple version complete!\n";

    //Blocked Version
    begin_roi();
    convolution_layer_blocked(*synapse,*neuron_i,*neuron_n2);
    end_roi();
    cout << "blocked computation complete!\n";

    //cuda Version
    begin_roi();
    int N = 1<<20; //HOW MANY ELEMENTS??
    int blockSize = 1024;
    int numBlocks = (N + blockSize-1) / blockSize;
    conv_cu <<<numBlocks, blockSize>>> (*synapse_cu, *neuron_i_cu, *neuron_n_cu);
    hipDeviceSynchronize();
    end_roi();
    
    //store result into cuda_result
    cout << "storing cuda result\n";
    for(int i=0;i<NYSCL;i++)
    {
        for(int j=0;j<NXSCL;j++)
        {
            for(int out=0;out<Nn;out++)
            {
                (*cuda_result)[i][j][out] = (*neuron_n_cu)[out][i][j];
            }
        }
    }
    cout << "cuda complete!\n";
    
    //compare results
    compare((VTYPE*)*neuron_n,(VTYPE*)*neuron_n2,NYSCL*NXSCL*Nn);
    compare((VTYPE*)*neuron_n,(VTYPE*)*cuda_result,NYSCL*NXSCL*Nn);
    
    //free memory allocated for cuda
    hipFree(synapse_cu);
    hipFree(neuron_i_cu);
    hipFree(neuron_n_cu);
    
    cout << "done\n";
}



